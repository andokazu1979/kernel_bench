
#include <hip/hip_runtime.h>
#ifdef CUDA

#include "kernel.h" 
//#include "kernel_cuda.h" 

__global__ void add(int n_, double* v1, double* v2, double* v3) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n_; i += stride) {
    v3[i] = v1[i] + v2[i];
  }
}

__global__ void add2(int n_, double* v1, double* v2, double* v3) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < n_) v3[index] = v1[index] + v2[index];
}

void KernelAddCuda::init(int n) {
  cudaMallocManaged(&v1, n*sizeof(double)); 
  cudaMallocManaged(&v2, n*sizeof(double)); 
  cudaMallocManaged(&v3, n*sizeof(double)); 

  for(int i = 0; i < n; i++) {
    //v1[i] = (double)rand();
    v1[i] = (double)i;
    //v2[i] = (double)rand();
    v2[i] = (double)i;
    v3[i] = 0.0f;
  } 
}

void KernelAddCuda::calc(int n_, int nitr_) {
  for(int itr = 0; itr < nitr_; itr++) {
    add<<<1, 256>>>(n_, v1, v2, v3);
    cudaDeviceSynchronize();
    if(v3[n_-1] < 0) dummy(v3);
  }
}

void KernelAddCuda::fin(int n) {
  cudaFree(v1);
  cudaFree(v2);
  cudaFree(v3);
}

void KernelAddCuda2::init(int n) {
  cudaMallocManaged(&v1, n*sizeof(double)); 
  cudaMallocManaged(&v2, n*sizeof(double)); 
  cudaMallocManaged(&v3, n*sizeof(double)); 

  for(int i = 0; i < n; i++) {
    v1[i] = (double)i;
    v2[i] = (double)i;
    v3[i] = 0.0;
  }
}

void KernelAddCuda2::calc(int n_, int nitr_) {
  dim3 block(n_);
  dim3 grid((n_ + block.x - 1) / block.x);
  for(int itr = 0; itr < nitr_; itr++) {
    add2<<<grid, block>>>(n_, v1, v2, v3);
    cudaDeviceSynchronize();
    if(v3[n_-1] < 0) dummy(v3);
  }
}

void KernelAddCuda2::fin(int n) {
  cudaFree(v1);
  cudaFree(v2);
  cudaFree(v3);
}
#endif
